#include <iostream>
#include "common.h"

int main(void) {
    hipDeviceProp_t prop;
    int dev;
    hipError_t err;

    err = hipGetDevice(&dev);
    HANDLE_ERROR(err);

    std::cout<< "ID of current device: " << dev << std::endl;

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 7;
    prop.minor = 0;
    err = hipChooseDevice(&dev, &prop);
    HANDLE_ERROR(err);

    std::cout<< "ID of device closest to compute capability 7.0: " << dev << std::endl;

    err = hipSetDevice(dev);
    HANDLE_ERROR(err);

    return  0;
}