#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "cpu_bitmap.h"


#define DIM 1000

struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__global__ void kernel(unsigned char* ptr);
__device__ int julia(int x, int y);

int main(void) {
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* dev_bitmap;

    hipError_t err;
    err = hipMalloc((void**)&dev_bitmap, bitmap.image_size());
    HANDLE_ERROR(err);

    dim3 grid(DIM, DIM);
    kernel<<<grid, 1>>>(dev_bitmap);

    err = hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
    HANDLE_ERROR(err);

    bitmap.display_and_exit();
    hipFree(dev_bitmap);

    return 0;
}

__global__ void kernel(unsigned char* ptr) {
    // Map the threadIdx/BlockIdx to a pixel position.
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;
    
    int juliaValue = julia(x, y);
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

__device__ int julia(int x, int y) {
    const float scale = 1.5;
    float jx = scale * static_cast<float>((DIM/2 - x)) / (DIM/2);
    float jy = scale * static_cast<float>((DIM/2 - y)) / (DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    for (int i=0; i<200; i++) {
        a = a*a + c;
        if (a.magnitude2() > 1000) {
            return 0;
        }
    }
    return 1;
}