#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"

#define N 10000

__global__ void add(int* a, int* b, int* c) {
    int tid = blockIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    hipError_t err;

    // Allocate memory on the GPU.
    err = hipMalloc((void**)&dev_a, N*sizeof(int));
    HANDLE_ERROR(err);
    err = hipMalloc((void**)&dev_b, N*sizeof(int));
    HANDLE_ERROR(err);
    err = hipMalloc((void**)&dev_c, N*sizeof(int));

    // Fill in arrays.
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy arrays into the GPU.
    err = hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    HANDLE_ERROR(err);
    err = hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    HANDLE_ERROR(err);

    add<<<N,1>>>(dev_a, dev_b, dev_c);

    // Copy array c from GPU to host.
    err = hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
    HANDLE_ERROR(err);

    // Display results.
    for (int i=0; i<N; i++) {
        if (i < 10) {
            std::cout<< a[i] << " + " << b[i] << " = " << c[i] << std::endl; 
        }
    }

    // Free allocated memory.
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}